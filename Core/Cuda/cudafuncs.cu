#include "hip/hip_runtime.h"
/*
 * This file is part of ElasticFusion.
 *
 * Copyright (C) 2015 Imperial College London
 *
 * The use of the code within this file and all code within files that
 * make up the software that is ElasticFusion is permitted for
 * non-commercial purposes only.  The full terms and conditions that
 * apply to the code within this file are detailed within the LICENSE.txt
 * file and at
 * <http://www.imperial.ac.uk/dyson-robotics-lab/downloads/elastic-fusion/elastic-fusion-license/>
 * unless explicitly stated.  By downloading this file you agree to
 * comply with these terms.
 *
 * If you wish to use any of this code for commercial purposes then
 * please email researchcontracts.engineering@imperial.ac.uk.
 *
 * Software License Agreement (BSD License)
 *
 *  Copyright (c) 2011, Willow Garage, Inc.
 *  All rights reserved.
 *
 *  Redistribution and use in source and binary forms, with or without
 *  modification, are permitted provided that the following conditions
 *  are met:
 *
 *   * Redistributions of source code must retain the above copyright
 *     notice, this list of conditions and the following disclaimer.
 *   * Redistributions in binary form must reproduce the above
 *     copyright notice, this list of conditions and the following
 *     disclaimer in the documentation and/or other materials provided
 *     with the distribution.
 *   * Neither the name of Willow Garage, Inc. nor the names of its
 *     contributors may be used to endorse or promote products derived
 *     from this software without specific prior written permission.
 *
 *  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 *  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 *  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
 *  FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
 *  COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
 *  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 *  BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 *  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 *  CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 *  LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
 *  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 *  POSSIBILITY OF SUCH DAMAGE.
 *
 *  Author: Anatoly Baskeheev, Itseez Ltd, (myname.mysurname@mycompany.com)
 */

#include "convenience.cuh"
#include "cudafuncs.cuh"
#include "operators.cuh"

inline hipError_t initTextureObjectFromArray(hipTextureObject_t * obj, hipArray_t cuArr)
{
  hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypeArray;
  resDesc.res.array.array = cuArr;

  hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.normalizedCoords = false;
  texDesc.filterMode = hipFilterModePoint;
  texDesc.addressMode[0] = hipAddressModeClamp;
  texDesc.addressMode[1] = hipAddressModeClamp;
  texDesc.readMode = hipReadModeElementType;

  return hipCreateTextureObject(obj, &resDesc, &texDesc, NULL);
}

__global__ void
pyrDownGaussKernel(const PtrStepSz<uint16_t> src, PtrStepSz<uint16_t> dst, float sigma_color) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= dst.cols || y >= dst.rows)
    return;

  const int D = 5;

  int center = src.ptr(2 * y)[2 * x];

  int x_mi = max(0, 2 * x - D / 2) - 2 * x;
  int y_mi = max(0, 2 * y - D / 2) - 2 * y;

  int x_ma = min(src.cols, 2 * x - D / 2 + D) - 2 * x;
  int y_ma = min(src.rows, 2 * y - D / 2 + D) - 2 * y;

  float sum = 0;
  float wall = 0;

  float weights[] = {0.375f, 0.25f, 0.0625f};

  for (int yi = y_mi; yi < y_ma; ++yi)
    for (int xi = x_mi; xi < x_ma; ++xi) {
      int val = src.ptr(2 * y + yi)[2 * x + xi];

      if (abs(val - center) < 3 * sigma_color) {
        sum += val * weights[abs(xi)] * weights[abs(yi)];
        wall += weights[abs(xi)] * weights[abs(yi)];
      }
    }

  dst.ptr(y)[x] = static_cast<int>(sum / wall);
}

void pyrDown(const DeviceArray2D<uint16_t>& src, DeviceArray2D<uint16_t>& dst) {
  dst.create(src.rows() / 2, src.cols() / 2);

  dim3 block(32, 8);
  dim3 grid(getGridDim(dst.cols(), block.x), getGridDim(dst.rows(), block.y));

  const float sigma_color = 30;

  pyrDownGaussKernel<<<grid, block>>>(src, dst, sigma_color);
  cudaSafeCall(hipGetLastError());
}

__global__ void computeVmapKernel(
    const PtrStepSz<uint16_t> depth,
    PtrStep<float> vmap,
    float fx_inv,
    float fy_inv,
    float cx,
    float cy,
    float depthCutoff) {
  int u = threadIdx.x + blockIdx.x * blockDim.x;
  int v = threadIdx.y + blockIdx.y * blockDim.y;

  if (u < depth.cols && v < depth.rows) {
    float z = depth.ptr(v)[u] / 1000.f; // load and convert: mm -> meters

    if (z != 0 && z < depthCutoff) {
      float vx = z * (u - cx) * fx_inv;
      float vy = z * (v - cy) * fy_inv;
      float vz = z;

      vmap.ptr(v)[u] = vx;
      vmap.ptr(v + depth.rows)[u] = vy;
      vmap.ptr(v + depth.rows * 2)[u] = vz;
    } else {
      vmap.ptr(v)[u] = __int_as_float(0x7fffffff); /*HIP_NAN_F*/
    }
  }
}

void createVMap(
    const CameraModel& intr,
    const DeviceArray2D<uint16_t>& depth,
    DeviceArray2D<float>& vmap,
    const float depthCutoff) {
  vmap.create(depth.rows() * 3, depth.cols());

  dim3 block(32, 8);
  dim3 grid(1, 1, 1);
  grid.x = getGridDim(depth.cols(), block.x);
  grid.y = getGridDim(depth.rows(), block.y);

  float fx = intr.fx, cx = intr.cx;
  float fy = intr.fy, cy = intr.cy;

  computeVmapKernel<<<grid, block>>>(depth, vmap, 1.f / fx, 1.f / fy, cx, cy, depthCutoff);
  cudaSafeCall(hipGetLastError());
}

__global__ void
computeNmapKernel(int rows, int cols, const PtrStep<float> vmap, PtrStep<float> nmap) {
  int u = threadIdx.x + blockIdx.x * blockDim.x;
  int v = threadIdx.y + blockIdx.y * blockDim.y;

  if (u >= cols || v >= rows)
    return;

  if (u == cols - 1 || v == rows - 1) {
    nmap.ptr(v)[u] = __int_as_float(0x7fffffff); /*HIP_NAN_F*/
    return;
  }

  float3 v00, v01, v10;
  v00.x = vmap.ptr(v)[u];
  v01.x = vmap.ptr(v)[u + 1];
  v10.x = vmap.ptr(v + 1)[u];

  if (!isnan(v00.x) && !isnan(v01.x) && !isnan(v10.x)) {
    v00.y = vmap.ptr(v + rows)[u];
    v01.y = vmap.ptr(v + rows)[u + 1];
    v10.y = vmap.ptr(v + 1 + rows)[u];

    v00.z = vmap.ptr(v + 2 * rows)[u];
    v01.z = vmap.ptr(v + 2 * rows)[u + 1];
    v10.z = vmap.ptr(v + 1 + 2 * rows)[u];

    float3 r = normalized(cross(v01 - v00, v10 - v00));

    nmap.ptr(v)[u] = r.x;
    nmap.ptr(v + rows)[u] = r.y;
    nmap.ptr(v + 2 * rows)[u] = r.z;
  } else
    nmap.ptr(v)[u] = __int_as_float(0x7fffffff); /*HIP_NAN_F*/
}

void createNMap(const DeviceArray2D<float>& vmap, DeviceArray2D<float>& nmap) {
  nmap.create(vmap.rows(), vmap.cols());

  int rows = vmap.rows() / 3;
  int cols = vmap.cols();

  dim3 block(32, 8);
  dim3 grid(1, 1, 1);
  grid.x = getGridDim(cols, block.x);
  grid.y = getGridDim(rows, block.y);

  computeNmapKernel<<<grid, block>>>(rows, cols, vmap, nmap);
  cudaSafeCall(hipGetLastError());
}

__global__ void tranformMapsKernel(
    int rows,
    int cols,
    const PtrStep<float> vmap_src,
    const PtrStep<float> nmap_src,
    const mat33 Rmat,
    const float3 tvec,
    PtrStepSz<float> vmap_dst,
    PtrStep<float> nmap_dst) {
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  if (x < cols && y < rows) {
    // vertexes
    float3 vsrc,
        vdst = make_float3(
            __int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff));
    vsrc.x = vmap_src.ptr(y)[x];

    if (!isnan(vsrc.x)) {
      vsrc.y = vmap_src.ptr(y + rows)[x];
      vsrc.z = vmap_src.ptr(y + 2 * rows)[x];

      vdst = Rmat * vsrc + tvec;

      vmap_dst.ptr(y + rows)[x] = vdst.y;
      vmap_dst.ptr(y + 2 * rows)[x] = vdst.z;
    }

    vmap_dst.ptr(y)[x] = vdst.x;

    // normals
    float3 nsrc,
        ndst = make_float3(
            __int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff));
    nsrc.x = nmap_src.ptr(y)[x];

    if (!isnan(nsrc.x)) {
      nsrc.y = nmap_src.ptr(y + rows)[x];
      nsrc.z = nmap_src.ptr(y + 2 * rows)[x];

      ndst = Rmat * nsrc;

      nmap_dst.ptr(y + rows)[x] = ndst.y;
      nmap_dst.ptr(y + 2 * rows)[x] = ndst.z;
    }

    nmap_dst.ptr(y)[x] = ndst.x;
  }
}

void tranformMaps(
    const DeviceArray2D<float>& vmap_src,
    const DeviceArray2D<float>& nmap_src,
    const mat33& Rmat,
    const float3& tvec,
    DeviceArray2D<float>& vmap_dst,
    DeviceArray2D<float>& nmap_dst) {
  int cols = vmap_src.cols();
  int rows = vmap_src.rows() / 3;

  vmap_dst.create(rows * 3, cols);
  nmap_dst.create(rows * 3, cols);

  dim3 block(32, 8);
  dim3 grid(1, 1, 1);
  grid.x = getGridDim(cols, block.x);
  grid.y = getGridDim(rows, block.y);

  tranformMapsKernel<<<grid, block>>>(
      rows, cols, vmap_src, nmap_src, Rmat, tvec, vmap_dst, nmap_dst);
  cudaSafeCall(hipGetLastError());
}

__global__ void copyMapsKernelTex(
    int rows,
    int cols,
    float* vmaps_tmp,
    PtrStepSz<float> vmap_dst,
    PtrStep<float> nmap_dst,
    hipTextureObject_t float4Tex0,
    hipTextureObject_t float4Tex1) {
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  if (x < cols && y < rows) {
    // vertexes
    float3 vsrc,
        vdst = make_float3(
            __int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff));

    float4 vmap_src = tex2D<float4>(float4Tex0, x, y);

    vmaps_tmp[y * cols * 4 + (x * 4) + 0] = vmap_src.x;
    vmaps_tmp[y * cols * 4 + (x * 4) + 1] = vmap_src.y;
    vmaps_tmp[y * cols * 4 + (x * 4) + 2] = vmap_src.z;
    vmaps_tmp[y * cols * 4 + (x * 4) + 3] = vmap_src.w;

    vsrc.x = vmap_src.x;
    vsrc.y = vmap_src.y;
    vsrc.z = vmap_src.z;

    if (!(vsrc.z == 0)) {
      vdst = vsrc;
    }

    vmap_dst.ptr(y)[x] = vdst.x;
    vmap_dst.ptr(y + rows)[x] = vdst.y;
    vmap_dst.ptr(y + 2 * rows)[x] = vdst.z;

    // normals
    float3 nsrc,
        ndst = make_float3(
            __int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff));

    float4 nmap_src = tex2D<float4>(float4Tex1, x, y);

    nsrc.x = nmap_src.x;
    nsrc.y = nmap_src.y;
    nsrc.z = nmap_src.z;

    if (!(vsrc.z == 0)) {
      ndst = nsrc;
    }

    nmap_dst.ptr(y)[x] = ndst.x;
    nmap_dst.ptr(y + rows)[x] = ndst.y;
    nmap_dst.ptr(y + 2 * rows)[x] = ndst.z;
  }
}

void copyMaps(
    const hipArray_t& vmap_src,
    const hipArray_t& nmap_src,
    const size_t srcWidth,
    const size_t srcHeight,
    DeviceArray<float>& vmaps_tmp,
    DeviceArray2D<float>& vmap_dst,
    DeviceArray2D<float>& nmap_dst) {
  vmap_dst.create(srcHeight * 3, srcWidth);
  nmap_dst.create(srcHeight * 3, srcWidth);

  dim3 block(32, 8);
  dim3 grid(1, 1, 1);
  grid.x = getGridDim(srcWidth, block.x);
  grid.y = getGridDim(srcHeight, block.y);

  hipTextureObject_t float4Tex0;
  cudaSafeCall(initTextureObjectFromArray(&float4Tex0, vmap_src));

  hipTextureObject_t float4Tex1;
  cudaSafeCall(initTextureObjectFromArray(&float4Tex1, nmap_src));

  copyMapsKernelTex<<<grid, block>>>(srcHeight, srcWidth, vmaps_tmp, vmap_dst, nmap_dst, float4Tex0, float4Tex1);
  cudaSafeCall(hipGetLastError());

  cudaSafeCall(hipDestroyTextureObject(float4Tex0));
  cudaSafeCall(hipDestroyTextureObject(float4Tex1));

  cudaSafeCall(hipGetLastError());
}

__global__ void copyVmapsTmpKernel(
    PtrStepSz<float> vmap_src,
    int rows,
    int cols,
    float* vmaps_tmp) {

  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  
  if (x < cols && y < rows) {
    vmaps_tmp[y * cols * 4 + (x * 4) + 2] = vmap_src.ptr(y + 2 * rows)[x];
  }
}

void copyVmapsTmp(
    DeviceArray2D<float>& vmap_src,
    const size_t srcWidth,
    const size_t srcHeight,
    DeviceArray<float>& vmaps_tmp) {
  dim3 block(32, 8);
  dim3 grid(1, 1, 1);
  grid.x = getGridDim(srcWidth, block.x);
  grid.y = getGridDim(srcHeight, block.y);
  
  copyVmapsTmpKernel<<<grid, block>>>(vmap_src, srcHeight, srcWidth, vmaps_tmp);
}

__global__ void
pyrDownKernelGaussF(const PtrStepSz<float> src, PtrStepSz<float> dst, float* gaussKernel) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= dst.cols || y >= dst.rows)
    return;

  const int D = 5;

  float center = src.ptr(2 * y)[2 * x];

  int tx = min(2 * x - D / 2 + D, src.cols - 1);
  int ty = min(2 * y - D / 2 + D, src.rows - 1);
  int cy = max(0, 2 * y - D / 2);

  float sum = 0;
  int count = 0;

  for (; cy < ty; ++cy) {
    for (int cx = max(0, 2 * x - D / 2); cx < tx; ++cx) {
      if (!isnan(src.ptr(cy)[cx])) {
        sum += src.ptr(cy)[cx] * gaussKernel[(ty - cy - 1) * 5 + (tx - cx - 1)];
        count += gaussKernel[(ty - cy - 1) * 5 + (tx - cx - 1)];
      }
    }
  }
  dst.ptr(y)[x] = (float)(sum / (float)count);
}

template <bool normalize>
__global__ void resizeMapKernel(
    int drows,
    int dcols,
    int srows,
    const PtrStep<float> input,
    PtrStep<float> output) {
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  if (x >= dcols || y >= drows)
    return;

  const float qnan = __int_as_float(0x7fffffff);

  int xs = x * 2;
  int ys = y * 2;

  float x00 = input.ptr(ys + 0)[xs + 0];
  float x01 = input.ptr(ys + 0)[xs + 1];
  float x10 = input.ptr(ys + 1)[xs + 0];
  float x11 = input.ptr(ys + 1)[xs + 1];

  if (isnan(x00) || isnan(x01) || isnan(x10) || isnan(x11)) {
    output.ptr(y)[x] = qnan;
    return;
  } else {
    float3 n;

    n.x = (x00 + x01 + x10 + x11) / 4;

    float y00 = input.ptr(ys + srows + 0)[xs + 0];
    float y01 = input.ptr(ys + srows + 0)[xs + 1];
    float y10 = input.ptr(ys + srows + 1)[xs + 0];
    float y11 = input.ptr(ys + srows + 1)[xs + 1];

    n.y = (y00 + y01 + y10 + y11) / 4;

    float z00 = input.ptr(ys + 2 * srows + 0)[xs + 0];
    float z01 = input.ptr(ys + 2 * srows + 0)[xs + 1];
    float z10 = input.ptr(ys + 2 * srows + 1)[xs + 0];
    float z11 = input.ptr(ys + 2 * srows + 1)[xs + 1];

    n.z = (z00 + z01 + z10 + z11) / 4;

    if (normalize)
      n = normalized(n);

    output.ptr(y)[x] = n.x;
    output.ptr(y + drows)[x] = n.y;
    output.ptr(y + 2 * drows)[x] = n.z;
  }
}

template <bool normalize>
void resizeMap(const DeviceArray2D<float>& input, DeviceArray2D<float>& output) {
  int in_cols = input.cols();
  int in_rows = input.rows() / 3;

  int out_cols = in_cols / 2;
  int out_rows = in_rows / 2;

  output.create(out_rows * 3, out_cols);

  dim3 block(32, 8);
  dim3 grid(getGridDim(out_cols, block.x), getGridDim(out_rows, block.y));
  resizeMapKernel<normalize><<<grid, block>>>(out_rows, out_cols, in_rows, input, output);
  cudaSafeCall(hipGetLastError());
  cudaSafeCall(hipDeviceSynchronize());
}

void resizeVMap(const DeviceArray2D<float>& input, DeviceArray2D<float>& output) {
  resizeMap<false>(input, output);
}

void resizeNMap(const DeviceArray2D<float>& input, DeviceArray2D<float>& output) {
  resizeMap<true>(input, output);
}

void pyrDownGaussF(const DeviceArray2D<float>& src, DeviceArray2D<float>& dst) {
  dst.create(src.rows() / 2, src.cols() / 2);

  dim3 block(32, 8);
  dim3 grid(getGridDim(dst.cols(), block.x), getGridDim(dst.rows(), block.y));

  const float gaussKernel[25] = {1,  4, 6, 4,  1,  4,  16, 24, 16, 4, 6, 24, 36,
                                 24, 6, 4, 16, 24, 16, 4,  1,  4,  6, 4, 1};

  float* gauss_cuda;

  hipMalloc((void**)&gauss_cuda, sizeof(float) * 25);
  hipMemcpy(gauss_cuda, &gaussKernel[0], sizeof(float) * 25, hipMemcpyHostToDevice);

  pyrDownKernelGaussF<<<grid, block>>>(src, dst, gauss_cuda);
  cudaSafeCall(hipGetLastError());

  hipFree(gauss_cuda);
}

__global__ void pyrDownKernelIntensityGauss(
    const PtrStepSz<uint8_t> src,
    PtrStepSz<uint8_t> dst,
    float* gaussKernel) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= dst.cols || y >= dst.rows)
    return;

  const int D = 5;

  int center = src.ptr(2 * y)[2 * x];

  int tx = min(2 * x - D / 2 + D, src.cols - 1);
  int ty = min(2 * y - D / 2 + D, src.rows - 1);
  int cy = max(0, 2 * y - D / 2);

  float sum = 0;
  int count = 0;

  for (; cy < ty; ++cy)
    for (int cx = max(0, 2 * x - D / 2); cx < tx; ++cx) {
      // This might not be right, but it stops incomplete model images from making up colors
      if (src.ptr(cy)[cx] > 0) {
        sum += src.ptr(cy)[cx] * gaussKernel[(ty - cy - 1) * 5 + (tx - cx - 1)];
        count += gaussKernel[(ty - cy - 1) * 5 + (tx - cx - 1)];
      }
    }
  dst.ptr(y)[x] = (sum / (float)count);
}

void pyrDownUcharGauss(const DeviceArray2D<uint8_t>& src, DeviceArray2D<uint8_t>& dst) {
  dst.create(src.rows() / 2, src.cols() / 2);

  dim3 block(32, 8);
  dim3 grid(getGridDim(dst.cols(), block.x), getGridDim(dst.rows(), block.y));

  const float gaussKernel[25] = {1,  4, 6, 4,  1,  4,  16, 24, 16, 4, 6, 24, 36,
                                 24, 6, 4, 16, 24, 16, 4,  1,  4,  6, 4, 1};

  float* gauss_cuda;

  hipMalloc((void**)&gauss_cuda, sizeof(float) * 25);
  hipMemcpy(gauss_cuda, &gaussKernel[0], sizeof(float) * 25, hipMemcpyHostToDevice);

  pyrDownKernelIntensityGauss<<<grid, block>>>(src, dst, gauss_cuda);
  cudaSafeCall(hipGetLastError());

  hipFree(gauss_cuda);
}

__global__ void verticesToDepthKernel(const float* vmap_src, PtrStepSz<float> dst, float cutOff) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= dst.cols || y >= dst.rows)
    return;

  float z = vmap_src[y * dst.cols * 4 + (x * 4) + 2];

  dst.ptr(y)[x] = z > cutOff || z <= 0 ? __int_as_float(0x7fffffff) /*HIP_NAN_F*/ : z;
}

void verticesToDepth(DeviceArray<float>& vmap_src, DeviceArray2D<float>& dst, float cutOff) {
  dim3 block(32, 8);
  dim3 grid(getGridDim(dst.cols(), block.x), getGridDim(dst.rows(), block.y));

  verticesToDepthKernel<<<grid, block>>>(vmap_src, dst, cutOff);
  cudaSafeCall(hipGetLastError());
}

__global__ void bgr2IntensityKernel(PtrStepSz<uint8_t> dst, hipTextureObject_t uchar4Tex) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= dst.cols || y >= dst.rows)
    return;

  uchar4 src = tex2D<uchar4>(uchar4Tex, x, y);

  int value = (float)src.x * 0.114f + (float)src.y * 0.299f + (float)src.z * 0.587f;

  dst.ptr(y)[x] = value;
}

void imageBGRToIntensity(hipArray_t cuArr, DeviceArray2D<uint8_t>& dst) {
  dim3 block(32, 8);
  dim3 grid(getGridDim(dst.cols(), block.x), getGridDim(dst.rows(), block.y));

  hipTextureObject_t uchar4Tex;
  cudaSafeCall(initTextureObjectFromArray(&uchar4Tex, cuArr));

  bgr2IntensityKernel<<<grid, block>>>(dst, uchar4Tex);

  cudaSafeCall(hipGetLastError());

  cudaSafeCall(hipDestroyTextureObject(uchar4Tex));
}

__constant__ float gsobel_x3x3[9];
__constant__ float gsobel_y3x3[9];

__global__ void
applyKernel(const PtrStepSz<uint8_t> src, PtrStep<int16_t> dx, PtrStep<int16_t> dy) {
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  if (x >= src.cols || y >= src.rows)
    return;

  float dxVal = 0;
  float dyVal = 0;

  int kernelIndex = 8;
  for (int j = max(y - 1, 0); j <= min(y + 1, src.rows - 1); j++) {
    for (int i = max(x - 1, 0); i <= min(x + 1, src.cols - 1); i++) {
      dxVal += (float)src.ptr(j)[i] * gsobel_x3x3[kernelIndex];
      dyVal += (float)src.ptr(j)[i] * gsobel_y3x3[kernelIndex];
      --kernelIndex;
    }
  }

  dx.ptr(y)[x] = dxVal;
  dy.ptr(y)[x] = dyVal;
}

void computeDerivativeImages(
    DeviceArray2D<uint8_t>& src,
    DeviceArray2D<int16_t>& dx,
    DeviceArray2D<int16_t>& dy) {
  static bool once = false;

  if (!once) {
    float gsx3x3[9] = {
        0.52201, 0.00000, -0.52201, 0.79451, -0.00000, -0.79451, 0.52201, 0.00000, -0.52201};

    float gsy3x3[9] = {
        0.52201, 0.79451, 0.52201, 0.00000, 0.00000, 0.00000, -0.52201, -0.79451, -0.52201};

    hipMemcpyToSymbol(HIP_SYMBOL(gsobel_x3x3), gsx3x3, sizeof(float) * 9);
    hipMemcpyToSymbol(HIP_SYMBOL(gsobel_y3x3), gsy3x3, sizeof(float) * 9);

    cudaSafeCall(hipGetLastError());
    cudaSafeCall(hipDeviceSynchronize());

    once = true;
  }

  dim3 block(32, 8);
  dim3 grid(getGridDim(src.cols(), block.x), getGridDim(src.rows(), block.y));

  applyKernel<<<grid, block>>>(src, dx, dy);

  cudaSafeCall(hipGetLastError());
  cudaSafeCall(hipDeviceSynchronize());
}

__global__ void projectPointsKernel(
    const PtrStepSz<float> depth,
    PtrStepSz<float3> cloud,
    const float invFx,
    const float invFy,
    const float cx,
    const float cy) {
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  if (x >= depth.cols || y >= depth.rows)
    return;

  float z = depth.ptr(y)[x];

  cloud.ptr(y)[x].x = (float)((x - cx) * z * invFx);
  cloud.ptr(y)[x].y = (float)((y - cy) * z * invFy);
  cloud.ptr(y)[x].z = z;
}

void projectToPointCloud(
    const DeviceArray2D<float>& depth,
    const DeviceArray2D<float3>& cloud,
    CameraModel& intrinsics,
    const int& level) {
  dim3 block(32, 8);
  dim3 grid(getGridDim(depth.cols(), block.x), getGridDim(depth.rows(), block.y));

  CameraModel intrinsicsLevel = intrinsics(level);

  projectPointsKernel<<<grid, block>>>(
      depth,
      cloud,
      1.0f / intrinsicsLevel.fx,
      1.0f / intrinsicsLevel.fy,
      intrinsicsLevel.cx,
      intrinsicsLevel.cy);
  cudaSafeCall(hipGetLastError());
  cudaSafeCall(hipDeviceSynchronize());
}
